#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define DATATYPE int
#define SMEMSIZE 2048
#define REP 128
//#define conflictnum 32

texture <int,1,hipReadModeElementType> texref1;
texture <int,1,hipReadModeElementType> texref2;
__global__ void texture_broadcast(double *time,DATATYPE *out,int its,int conflictnum)
{
	DATATYPE p,q=(threadIdx.x/conflictnum*conflictnum);
	double time_tmp=0.0;
	unsigned int start_time=0,stop_time=0;
	unsigned int i,j;
	for (i=0;i<its;i++)
	{
		__syncthreads();
		start_time=clock();
#pragma unroll
		for (j=0;j<REP;j++)
		{
			p=tex1Dfetch(texref1,q);
			q=tex1Dfetch(texref2,p);
		}
		stop_time=clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/REP/its;
	out[blockDim.x*blockIdx.x+threadIdx.x] = p+q;
	time[blockDim.x*blockIdx.x+threadIdx.x] = time_tmp;
}

int main_test(int blocks,int threads,DATATYPE *h_in1,DATATYPE *h_in2, int conflictnum)
{
	int its=30;
	DATATYPE *d_in1,*d_in2;
	hipMalloc((void**)&d_in1,sizeof(DATATYPE)*SMEMSIZE);
	hipMalloc((void**)&d_in2,sizeof(DATATYPE)*SMEMSIZE);
	hipMemcpy(d_in1,h_in1,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_in2,h_in2,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	hipBindTexture(NULL,texref1,d_in1,sizeof(DATATYPE)*SMEMSIZE);
	hipBindTexture(NULL,texref2,d_in2,sizeof(DATATYPE)*SMEMSIZE);
	double *h_time,*d_time;
	DATATYPE *d_out;
	h_time=(double*)malloc(sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_time,sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_out,sizeof(DATATYPE)*blocks*threads);

	texture_broadcast<<<blocks,threads>>>(d_time,d_out,its,conflictnum);
	hipMemcpy(h_time,d_time,sizeof(double)*blocks*threads,hipMemcpyDeviceToHost);
	double avert=0.0,maxt=0.0,mint=99999.9;
	int nn=0;
	for (int i=0;i<blocks;i++)
	{
		for (int j=0;j<threads;j+=32)
		{
			avert+=h_time[i*threads+j];
			nn++;
			if (maxt<h_time[i*threads+j])
			{
				maxt=h_time[i*threads+j];
			}
			if (mint>h_time[i*threads+j])
			{
				mint=h_time[i*threads+j];
			}
		}
	}
	avert/=nn;
	printf("%d\t%d\t\t%f\t%f\t%f\n",blocks,threads,avert,mint,maxt);
	hipUnbindTexture(texref1);
	hipUnbindTexture(texref2);
	hipFree(d_time);
	hipFree(d_out);
	hipFree(d_in1);
	hipFree(d_in2);
	free(h_time);
	return 0;
}
void init_order(DATATYPE *a,int n)
{
	for (int i=0;i<n;i++)
	{
		a[i]=i;
	}
}

int main(int argc, char* argv[])
{
	if (argc != 2) {
        printf("%s <conflict> \n", argv[0]);
    } else {

    	int value = atoi(argv[1]);

		DATATYPE *h_in1;
		h_in1 = (DATATYPE*) malloc(sizeof(DATATYPE) * SMEMSIZE);

		init_order(h_in1,SMEMSIZE);

		printf("conflictnum\tblocks\tthreads\taver\tmin\tmax\t(clocks)\n");

		int blocks = 1;
		for (int j = 0; j <= 512; j += 32) {
			int threads = (j == 0 ? 1 : j);
			main_test(blocks, threads, h_in1, h_in1, value);
		}

		free(h_in1);
	}
	return 0;
}