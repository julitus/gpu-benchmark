#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#define DATATYPE int
#define ARRAYLEN 2048
#define REP 128
//#define PRINTNEED
#define TIMETESTEVENT
#include <hip/hip_runtime.h>
#include "repeat.h"

__global__ void test_register_latency(double *time,DATATYPE *out,int its)
{
	int p=3;
	int q=1;
	int r,x=2,y=5,z=7;
	double time_tmp=0.0;
	unsigned int start_time=0, stop_time=0;

	for (int i=0;i<its;i++)										
	{									
		__syncthreads();
		start_time = clock();	
		repeat128(r=p;p=q;q=x;x=y;y=z;z=r;)
		stop_time = clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/128.0/its;
	out[0] =r;
	time[0] = time_tmp;
}
__constant__ DATATYPE d_const_array[ARRAYLEN];
__global__ void test_const_latency(double *time,DATATYPE *out,int its)
{
	int p=0;
	double time_tmp=0.0;
	unsigned int start_time=0, stop_time=0;

	for (int i=0;i<its;i++)										
	{									
		__syncthreads();
		start_time = clock();	
		repeat128(p=d_const_array[p];)
		stop_time = clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/128.0/its;
	out[1] =p;
	time[1] = time_tmp;
}
__global__ void test_shared_latency(double *time,DATATYPE *out,int its,DATATYPE *array)
{
	__shared__ DATATYPE shared_array[ARRAYLEN];
	int i;
	for (i=0;i<ARRAYLEN;i++)
	{
		shared_array[i]=array[i];
	}
	int p=0;
	double time_tmp=0.0;
	unsigned int start_time=0, stop_time=0;

	for (int i=0;i<its;i++)										
	{									
		__syncthreads();
		start_time = clock();	
		repeat128(p=shared_array[p];)
		stop_time = clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/128.0/its;
	out[2] =p;
	time[2] = time_tmp;
}
__global__ void test_local_latency(double *time,DATATYPE *out,int its,DATATYPE *array)
{
	DATATYPE local_array[ARRAYLEN];
	int i;
	for (i=0;i<ARRAYLEN;i++)
	{
		local_array[i]=array[i];
	}
	int p=0;
	double time_tmp=0.0;
	unsigned int start_time=0, stop_time=0;

	for (int i=0;i<its;i++)										
	{									
		__syncthreads();
		start_time = clock();	
		repeat128(p=local_array[p];)
			stop_time = clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/128.0/its;
	out[3] =p;
	time[3] = time_tmp;
}
__global__ void test_global_latency(double *time,DATATYPE *out,int its,DATATYPE *array)
{
	int p=0;
	double time_tmp=0.0;
	unsigned int start_time=0, stop_time=0;

	for (int i=0;i<its;i++)										
	{									
		__syncthreads();
		start_time = clock();	
		repeat128(p=array[p];)
		stop_time = clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/128.0/its;
	out[4] =p;
	time[4] = time_tmp;
}

texture <int,1,hipReadModeElementType> texref;
__global__ void test_texture_latency(double *time,DATATYPE *out,int its)
{
	int p=0;
	double time_tmp=0.0;
	unsigned int start_time=0, stop_time=0;

	for (int i=0;i<its;i++)										
	{									
		__syncthreads();
		start_time = clock();	
		repeat128(p=tex1Dfetch(texref,p);)
		stop_time = clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/128.0/its;
	out[5] =p;
	time[5] = time_tmp;
}



void call_test_latency(int step,int its,double *h_time)
{
	DATATYPE *h_array;
	h_array=(DATATYPE*)malloc(sizeof(DATATYPE)*ARRAYLEN);
	for (int i=0;i<ARRAYLEN;i++)
	{
		h_array[i]=(i+step)%ARRAYLEN;
	}
	DATATYPE *d_array;
	hipMalloc((void**)&d_array,sizeof(DATATYPE)*ARRAYLEN);
	hipMemcpy(d_array,h_array,sizeof(DATATYPE)*ARRAYLEN,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_const_array),h_array,sizeof(DATATYPE)*ARRAYLEN);

	/*texture*/
	hipBindTexture(NULL,texref,d_array,sizeof(DATATYPE)*ARRAYLEN);

	double *d_time;
	hipMalloc((void**)&d_time,sizeof(double)*6);
	DATATYPE *d_out,*h_out;
	h_out=(DATATYPE *)malloc(sizeof(DATATYPE)*6);
	hipMalloc((void**)&d_out,sizeof(DATATYPE)*6);

	test_register_latency	<<<1,1>>>(d_time,d_out,its);
	test_const_latency		<<<1,1>>>(d_time,d_out,its);
	test_shared_latency		<<<1,1>>>(d_time,d_out,its,d_array);
	test_local_latency		<<<1,1>>>(d_time,d_out,its,d_array);
	test_global_latency		<<<1,1>>>(d_time,d_out,its,d_array);
	test_texture_latency	<<<1,1>>>(d_time,d_out,its);

	hipMemcpy(h_out,d_out,sizeof(DATATYPE)*6,hipMemcpyDeviceToHost);
	hipMemcpy(h_time,d_time,sizeof(double)*6,hipMemcpyDeviceToHost);
	printf("%d:\t%f\t%f\t%f\t%f\t%f\t%f\n",step,h_time[0],h_time[1],h_time[2],h_time[3],h_time[4],h_time[5]);
//	printf("out=%d\t%d\t%d\t%d\t%d\n",h_out[0],h_out[1],h_out[2],h_out[3],h_out[4]);

	hipUnbindTexture(texref);
	hipFree(d_array);
	hipFree(d_time);
	hipFree(d_out);
	free(h_array);
	free(h_out);
}


int  main()
{
	double *h_time;
	h_time = (double *) malloc(sizeof(double) * 6 * 1024);
	printf("step:register\t constant\t shared\t local\t global\t texture\n");
	
	for (int i = 1; i <= 1024; i++) {
		call_test_latency(i, 30, &h_time[(i - 1) * 6]);
	}

	printf("average:\t");
	for (int i = 0; i < 6; i++) {
		double average = 0.0;
		for (int j = 0; j < 1024; j++) {
			average += h_time[j * 6 + i];
		}
		average /= 1024.0;
		printf("%f\t", average);
	}
	printf("\n");
	return 0;
}