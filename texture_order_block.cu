#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define DATATYPE int
#define SMEMSIZE 512
#define REP 128

texture <int,1,hipReadModeElementType> texref1;
texture <int,1,hipReadModeElementType> texref2;
__global__ void texture_order_1(double *time,DATATYPE *out,int its)
{
	DATATYPE p,q=threadIdx.x;
	double time_tmp=0.0;
	unsigned int start_time=0,stop_time=0;
	unsigned int i,j;
	for (i=0;i<its;i++)
	{
		__syncthreads();
		start_time=clock();
#pragma unroll
		for (j=0;j<REP;j++)
		{
			p=tex1Dfetch(texref1,q);
			q=tex1Dfetch(texref2,p);
		}
		stop_time=clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/REP/its;
	out[blockDim.x*blockIdx.x+threadIdx.x] = p+q;
	time[blockDim.x*blockIdx.x+threadIdx.x] = time_tmp;
}

int main_test(int blocks,int threads,DATATYPE *h_in1,DATATYPE *h_in2,int xxx)
{
	int its=30;
	DATATYPE *d_in1,*d_in2;
	hipMalloc((void**)&d_in1,sizeof(DATATYPE)*SMEMSIZE);
	hipMalloc((void**)&d_in2,sizeof(DATATYPE)*SMEMSIZE);
	hipMemcpy(d_in1,h_in1,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_in2,h_in2,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	hipBindTexture(NULL,texref1,d_in1,sizeof(DATATYPE)*SMEMSIZE);
	hipBindTexture(NULL,texref2,d_in2,sizeof(DATATYPE)*SMEMSIZE);
	double *h_time,*d_time;
	DATATYPE *d_out;
	h_time=(double*)malloc(sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_time,sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_out,sizeof(DATATYPE)*blocks*threads);

	texture_order_1<<<blocks,threads>>>(d_time,d_out,its);
	hipMemcpy(h_time,d_time,sizeof(double)*blocks*threads,hipMemcpyDeviceToHost);
	double avert=0.0,maxt=0.0,mint=99999.9;
	int nn=0;
	for (int i=0;i<blocks;i++)
	{
		for (int j=0;j<threads;j+=32)
		{
			avert+=h_time[i*threads+j];
			nn++;
			if (maxt<h_time[i*threads+j])
			{
				maxt=h_time[i*threads+j];
			}
			if (mint>h_time[i*threads+j])
			{
				mint=h_time[i*threads+j];
			}
		}
	}
	avert/=nn;
	printf("%d\t%d\t%d\t\t%f\t%f\t%f\n",xxx, blocks,threads,avert,mint,maxt);
	hipUnbindTexture(texref1);
	hipUnbindTexture(texref2);
	hipFree(d_time);
	hipFree(d_out);
	hipFree(d_in1);
	hipFree(d_in2);
	free(h_time);
	return 0;
}
void init_order(DATATYPE *a,int n)
{
	for (int i=0;i<n;i++)
	{
		a[i]=i;
	}
}
void init_disordered_32(DATATYPE *a,int n)
{
	DATATYPE p[32];
	for (int i=0;i<32;i++)
	{
		p[i]=i;
	}
	for (int i=0;i<n;i+=32)
	{
		for (int j=0;j<32;j++)
		{
			int jj=rand()%(32-j);
			a[i+j]=p[jj];
			for (int k=jj;k<(32-j);k++)
			{
				p[k]=p[k+1];
			}
		}
		for (int j=0;j<32;j++)
		{
			p[j]=a[i+j];
			a[i+j]+=i;
		}
	}
}
void init_disordered_512(DATATYPE *a,int n)
{
	const int nn=n/32;
	DATATYPE *q=(DATATYPE*)malloc(sizeof(DATATYPE)*nn);
	DATATYPE *b=(DATATYPE*)malloc(sizeof(DATATYPE)*n);
	init_order(q,nn);

	for (int i=0;i<n;i+=nn)
	{
		for (int j=0;j<nn;j++)
		{
			int jj=rand()%(nn-j);
			b[i+j]=q[jj];
			for (int k=jj;k<(nn-j);k++)
			{
				q[k]=q[k+1];
			}
		}
		for (int j=0;j<nn;j++)
		{
			q[j]=b[i+j];
		}
	}

	DATATYPE p[32];
	for (int i=0;i<32;i++)
	{
		p[i]=i;
	}
	for (int i=0;i<32;i++)
	{
		for (int j=0;j<nn;j++)
		{
			a[j*32+i]=b[i*nn+j]*32+p[i];
		}
	}
	free(q);
	free(b);
}

int main()
{
	DATATYPE *h_in1, *h_in2, *h_in3;
	h_in1 = (DATATYPE *) malloc(sizeof(DATATYPE) * SMEMSIZE);
	h_in2 = (DATATYPE *) malloc(sizeof(DATATYPE) * SMEMSIZE);
	h_in3 = (DATATYPE *) malloc(sizeof(DATATYPE) * SMEMSIZE);
	init_order(h_in1, SMEMSIZE);
	init_disordered_32(h_in2, SMEMSIZE);
	init_disordered_512(h_in3, SMEMSIZE);

	printf("blocks\t threads\t aver \t min \t max \t(clocks)\n");

	for (int i = 0; i <= 1024; i += 32) {
		int blocks = (i == 0 ? 1 : i);
		int threads = 256;
		main_test(blocks, threads, h_in1, h_in1, 1);
		main_test(blocks, threads, h_in2, h_in2, 2);
		main_test(blocks, threads, h_in3, h_in3, 3);
	}

	free(h_in1);
	free(h_in2);
	free(h_in3);

	return 0;
}